
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void kernel() { printf("cuda kernel called!\n"); }

void launch() {
  kernel<<<3, 2>>>();
  hipDeviceSynchronize();
}


int main() {
  launch();
  return 0;
}
